#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timer.hpp"
 
 
__global__
void saxpy(int n, double a, double *x, double *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}
 
int main(void)
{
  int N[7];
  N[0] = 100;
  N[1] = 300;
  N[2] = 1000;
  N[3] = 10000;
  N[4] = 100000;
  N[5] = 1000000;
  N[6] = 3000000;
  N[7] = 0;

  for(int j=0; j < 7; j++){ 
  double *x, *y, *d_x, *d_y;
  Timer timer;
  double time = 0.0;
  double time_sum = 0.0;

  for(int it=0; it<10;it++){
 
  // Allocate device memory and copy host data over
  hipDeviceSynchronize();
  timer.reset();
  x = (double*)malloc(N[j]*sizeof(double));
  for (int i = 0; i < N[j]; i++) {
    x[i] = 1.0f;
  }
  hipMemcpy(d_x, x, N[j]*sizeof(double), hipMemcpyHostToDevice);
 
  time = timer.get();
  hipDeviceSynchronize();
  time_sum += time;
  free(x);
  hipFree(d_x);
  }
  printf("%f\n", time_sum/10);
}
  return EXIT_SUCCESS;
}
 