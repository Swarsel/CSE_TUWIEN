#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <ostream>
#include <stdio.h>
#include <cmath>
#include <iostream>
#include "timer.hpp"
#include <cuda_errchk.hpp>

__global__ void dot(double *x, double **y, double *results, int i, int N) {
    double alpha1{0}, alpha2{0}, alpha3{0}, alpha4{0}, alpha5{0}, alpha6{0}, alpha7{0}, alpha8{0};

    for(int j = blockIdx.x * blockDim.x + threadIdx.x; j < N; j += blockDim.x*gridDim.x) {
        double val_w = x[j];
        alpha1 += val_w * y[i][j];
        alpha2 += val_w * y[i+1][j];
        alpha3 += val_w * y[i+2][j];
        alpha4 += val_w * y[i+3][j];
        alpha5 += val_w * y[i+4][j];
        alpha6 += val_w * y[i+5][j];
        alpha7 += val_w * y[i+6][j];
        alpha8 += val_w * y[i+7][j];
    }

    for (int j=16; j>0; j=j/2) {
        alpha1 += __shfl_xor_sync(0xffffffff, alpha1, j);
        alpha2 += __shfl_xor_sync(0xffffffff, alpha2, j);
        alpha3 += __shfl_xor_sync(0xffffffff, alpha3, j);
        alpha4 += __shfl_xor_sync(0xffffffff, alpha4, j);
        alpha5 += __shfl_xor_sync(0xffffffff, alpha5, j);
        alpha6 += __shfl_xor_sync(0xffffffff, alpha6, j);
        alpha7 += __shfl_xor_sync(0xffffffff, alpha7, j);
        alpha8 += __shfl_xor_sync(0xffffffff, alpha8, j);
    }

    if (threadIdx.x % 32 == 0) {
        atomicAdd(&results[i], alpha1);
        atomicAdd(&results[i+1], alpha2);
        atomicAdd(&results[i+2], alpha3);
        atomicAdd(&results[i+3], alpha4);
        atomicAdd(&results[i+4], alpha5);
        atomicAdd(&results[i+5], alpha6);
        atomicAdd(&results[i+6], alpha7);
        atomicAdd(&results[i+7], alpha8);
    }
}

int main(int argc, char *argv[])
{
    double time;
    // const size_t N = 100000;
    // const size_t K = 16;
    Timer timer;
    // int Ns[5] = {10000,100000,1000000,10000000,20000000};
    const int N = std::atoi(argv[1]);
    const int K = std::atoi(argv[2]);
    // int Ks[4] = {8,16,24,32};
    // int K = std::atoi(argv[1]);
    // for (int ki=0; ki<=3; ki++) {
        // int K = Ks[ki];
        // std::cout << "K " << K << std::endl;
        // std::cout << "[";

    // for (int ni = 0; ni <= 4; ni++) {
            // int N = Ns[ni];

            //
            // Initialize CUBLAS:
            //
            // std::cout << "Init CUBLAS..." << std::endl;
            // hipblasHandle_t h;
            // hipblasCreate(&h);


            //
            // allocate host memory:
            //
            // std::cout << "Allocating host arrays..." << std::endl;
            double  *x = (double*)malloc(sizeof(double) * N);
            double **y = (double**)malloc(sizeof(double*) * K);
            for (size_t i=0; i<K; ++i) {
                y[i] = (double*)malloc(sizeof(double) * N);
            }
            double *results  = (double*)malloc(sizeof(double) * K);
            double *results2 = (double*)malloc(sizeof(double) * K);


            //
            // allocate device memory
            //
            // std::cout << "Allocating CUDA arrays..." << std::endl;
            double *cuda_x; hipMalloc(&cuda_x, sizeof(double)*N);
            double *cuda_results; hipMalloc(&cuda_results, sizeof(double)*K);
            double **cuda_y; hipMalloc(&cuda_y, sizeof(double*) * K);
            double **y_pointers = (double**)malloc(sizeof(double*) * K);
            for (size_t i=0; i<K; ++i) {
                hipMalloc(&y_pointers[i], sizeof(double)*N);
            }
            hipMemcpy(cuda_y,y_pointers,sizeof(double*) * K, hipMemcpyHostToDevice);

            //
            // fill host arrays with values
            //
            for (size_t j=0; j<N; ++j) {
                x[j] = 1 + j%K;
            }
            for (size_t i=0; i<K; ++i) {
                for (size_t j=0; j<N; ++j) {
                    y[i][j] = 1 + rand() / (1.1 * RAND_MAX);
                }
            }

            //
            // Reference calculation on CPU:
            //
            for (size_t i=0; i<K; ++i) {
                results[i] = 0;
                results2[i] = 0;
                for (size_t j=0; j<N; ++j) {
                    results[i] += x[j] * y[i][j];
                }
            }

            //
            // Copy data to GPU
            //
            // std::cout << "Copying data to GPU..." << std::endl;
            hipMemcpy(cuda_x, x, sizeof(double)*N, hipMemcpyHostToDevice);
            for (size_t i=0; i<K; ++i) {
                hipMemcpy(y_pointers[i], y[i], sizeof(double)*N, hipMemcpyHostToDevice);
            }
            time = 0;
            std::fill(results2, results2 + K, 0);
            for(int it=0; it < 10; it++) {
                hipMemcpy(cuda_results, results2, sizeof(double)*K, hipMemcpyHostToDevice);
                timer.reset();
                for (int i=0; i<=(K-8); i += 8) dot<<<256, 256>>>(cuda_x, cuda_y, cuda_results, i, N);
                CUDA_ERRCHK(hipDeviceSynchronize());
                time += timer.get();
            }
            time /= 10;
            std::cout << time;

            // if (N != Ns[4]) std::cout << time << ", ";
            // else if (N == Ns[4]) std::cout << time << "]" << std::endl;

            hipMemcpy(results2, cuda_results, sizeof(double) * K, hipMemcpyDeviceToHost);
            // Compare results
            //
            // std::cout << "Copying results back to host..." << std::endl;
            for (size_t i=0; i<K; ++i) {
                if (fabs(results[i] - results2[i]) / results[i] > 1e-10) {
                    std::cout << std::endl << "ATTENTION WRONG RESULT:" << results[i] << " on CPU, " << results2[i] << " on GPU. Relative difference: " << fabs(results[i] - results2[i]) / results[i] << std::endl;
                    return 1;
                }
            }

            //
            // Clean up:
            //
            // std::cout << "Cleaning up..." << std::endl;
            free(x);
            hipFree(cuda_x);

            for (size_t i=0; i<K; ++i) {
                free(y[i]);
                hipFree(y_pointers[i]);
            }
            free(y);
            hipFree(cuda_y);
            hipFree(cuda_results);

            free(results);
            free(results2);
            free(y_pointers);

            // hipblasDestroy(h);
        // }
    // }

    return 0;
}
