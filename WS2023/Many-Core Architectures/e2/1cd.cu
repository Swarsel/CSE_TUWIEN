#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timer.hpp"


__global__ void add(int n, double *x, double *y, double *z)
{
  unsigned int total_threads =  blockDim.x * gridDim.x;;
  int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
  for (int i = thread_id; i<n; i += total_threads) z[i] = x[i] + y[i];
}

int main(void)
{
  int N[7] = {100, 300, 1000, 10000, 100000, 1000000, 3000000};
  for(int j=0; j <= 6; j++){

    double *x, *y, *z, *d_x, *d_y, *d_z;
    Timer timer;
    double time_sum = 0.0;

    for(int it=0; it<10; it++){
      x = (double*)malloc(N[j]*sizeof(double));
      y = (double*)malloc(N[j]*sizeof(double));
      z = (double*)malloc(N[j]*sizeof(double));
      hipMalloc(&d_x, N[j]*sizeof(double));
      hipMalloc(&d_y, N[j]*sizeof(double));
      hipMalloc(&d_z, N[j]*sizeof(double));

      for (int i = 0; i < N[j]; i++) {
        x[i] = (double)i;
        y[i] = (double)(N[j]-i-1);
      }
      hipMemcpy(d_x, x, N[j]*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_y, y, N[j]*sizeof(double), hipMemcpyHostToDevice);
      hipDeviceSynchronize();

      timer.reset();
      add<<<256, 256>>>(N[j], d_x, d_y, d_z);
      hipDeviceSynchronize();
      time_sum += timer.get();

      hipFree(d_x);
      hipFree(d_y);
      hipFree(d_z);
      free(x);
      free(y);
      };
    printf("%f\n", time_sum/10);
    }
  return EXIT_SUCCESS;
}
