#include <stdio.h>
#include "timer.hpp"

int main(void)
{

  int N[7] = {100, 300, 1000, 10000, 100000, 1000000, 3000000};

  for(int j=0; j <= 6; j++){
    double *x, *y, *d_x, *d_y;
    Timer timer;
    double time = 0.0;
    double time_sum = 0.0;

    for(int it=0; it<10; it++){

      hipDeviceSynchronize();
      timer.reset();
      x = (double*)malloc(N[j]*sizeof(double));
      y = (double*)malloc(N[j]*sizeof(double));
      hipMalloc(&d_x, N[j]*sizeof(double));
      hipMalloc(&d_y, N[j]*sizeof(double));
      for (int i = 0; i < N[j]; i++) {
        x[i] = (double)i;
        y[i] = (double)(N[j]-i-1);
      }
      hipMemcpy(d_x, x, N[j]*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_y, y, N[j]*sizeof(double), hipMemcpyHostToDevice);
      time = timer.get();

      hipDeviceSynchronize();
      time_sum += time;
      free(x);
      hipFree(d_x);
    }
    printf("%f\n", time_sum/10);
  }
  return EXIT_SUCCESS;
}
