#include "timer.hpp"
#include "cuda_errchk.hpp"
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <iostream>

int main(void) {
        double *x, *cuda_x;
        Timer timer;
        double time_sum = 0.0;
        x = (double*)malloc(1*sizeof(double));
        hipMalloc(&cuda_x, 1*sizeof(double));
        x[0] = 0;
        hipMemset(cuda_x, 1, 1);
        for (int it=0; it < 10000; it++) hipMemcpy(cuda_x, x, sizeof(double), hipMemcpyHostToDevice);

        timer.reset();
        hipMemcpy(cuda_x, x, sizeof(double), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        time_sum = timer.get();
        std::cout << time_sum;
        exit(0);
}