#include "timer.hpp"
#include "cuda_errchk.hpp"
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <iostream>

int main(void) {
        double *x, *cuda_x;
        Timer timer;
        double time_sum = 0.0;
        x = (double*)malloc(1*sizeof(double));
        hipMalloc(&cuda_x, 1*sizeof(double));
        x[0] = 0;
        hipMemset(cuda_x, 1, 1);
        // for(int it=0; it < 10; it++){
            timer.reset();
            hipMemcpy(cuda_x, x, sizeof(double), hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            // time_sum += timer.get();
            time_sum = timer.get();
            // hipMemset(cuda_x, 1, 1);
        // }
        // std::cout << time_sum/10;
        std::cout << time_sum;
        exit(0);
}