#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
#include <algorithm>
#include "poisson2d.hpp"
#include "timer.hpp"

/** Computes y = A*x for a sparse matrix A in CSR format and vector x,y  */
__global__
void csr_matvec_product(int N, int *rowoffsets, int *colindices, double *values, double *x, double *y) {
    for (int row = blockDim.x * blockIdx.x + threadIdx.x; row < N; row += gridDim.x * blockDim.x) {
        double val = 0;
        for (int jj = rowoffsets[row]; jj < rowoffsets[row+1]; ++jj) {
            val += values[jj] * x[colindices[jj]];
        }
        y[row] = val;
    }
}


__global__ void dot(int N, double *x, double *y, double *results) {
    double alpha1{0};
    unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id == 0) *results = 0;
    for(int j = thread_id; j < N; j += blockDim.x*gridDim.x) {
        alpha1 += x[j] * y[j];
    }

    for (int j=warpSize/2; j>0; j=j/2) {
        alpha1 += __shfl_xor_sync(0xffffffff, alpha1, j);
    }

    if (threadIdx.x % warpSize == 0) {
        atomicAdd(results, alpha1);
    }
}

__global__
void xrIterate(int N, double *x, double *p, double *r, double *Ap, double mod) {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
            x[i] += mod * p[i];
            r[i] -= mod * Ap[i];
        }
}

__global__
void vecIterate(int N, double *out, double *in1, double *in2, double mod) {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
            out[i] = in1[i] + mod * in2[i];
        }
}

/** Implementation of the conjugate gradient algorithm.
 *
 *  The control flow is handled by the CPU.
 *  Only the individual operations (vector updates, dot products, sparse matrix-vector product) are transferred to CUDA kernels.
 *
 *  The temporary arrays p, r, and Ap need to be allocated on the GPU for use with CUDA.
 *  Modify as you see fit.
 */
void conjugate_gradient(size_t N,  // number of unknows
                        int *csr_rowoffsets, int *csr_colindices, double *csr_values,
                        double *rhs,
                        double *solution)
//, double *init_guess)   // feel free to add a nonzero initial guess as needed
{
    // clear solution vector (it may contain garbage values):
    std::fill(solution, solution + N, 0);

    // initialize work vectors:
    double *p = (double*)malloc(sizeof(double) * N);
    double *r = (double*)malloc(sizeof(double) * N);
    double *Ap = (double*)malloc(sizeof(double) * N);

    // CPU variables
    double rr{0};
    double alpha{0};
    double beta{0};

    // line 2: initialize r and p:
    std::copy(rhs, rhs+N, p);
    std::copy(rhs, rhs+N, r);

    // initialize variables for GPU
    int *cuda_csr_rowoffsets, *cuda_csr_colindices;
    double *cuda_csr_values, *cuda_p, *cuda_r, *cuda_Ap, *cuda_out, *cuda_solution;
    hipMalloc(&cuda_csr_rowoffsets, sizeof(int) * (N + 1));
    hipMalloc(&cuda_csr_colindices, sizeof(int) * 5 * N);
    hipMalloc(&cuda_csr_values, sizeof(double) * 5 * N);
    hipMalloc(&cuda_p, sizeof(double) * N);
    hipMalloc(&cuda_r, sizeof(double) * N);
    hipMalloc(&cuda_Ap, sizeof(double) * N);
    hipMalloc(&cuda_out, sizeof(double) * 1);
    hipMalloc(&cuda_solution, sizeof(double) * N);
    // hipMalloc(&cuda_rprp, sizeof(double) * 1);

    hipMemcpy(cuda_csr_rowoffsets, csr_rowoffsets, sizeof(int) * (N + 1), hipMemcpyHostToDevice);
    hipMemcpy(cuda_csr_colindices, csr_colindices, sizeof(int) * 5 * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_csr_values, csr_values, sizeof(double) * 5 * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_p, p, sizeof(double) * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_r, r, sizeof(double) * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_solution, solution, sizeof(double) * N, hipMemcpyHostToDevice);


    //    hipMemset(&cuda_rr, 0, 1);
    dot<<<256,256>>>(N, cuda_r, cuda_r, cuda_out);
    // hipDeviceSynchronize();
    hipMemcpy(&rr, cuda_out, sizeof(double), hipMemcpyDeviceToHost);
    // std::cout << rr << std::endl;
    double rr0 = rr;

    int iters = 0;
    while (1) {

        // line 4: A*p:
        csr_matvec_product<<<128, 128>>>(N, cuda_csr_rowoffsets, cuda_csr_colindices,
                                                   cuda_csr_values, cuda_p, cuda_Ap);

        dot<<<128, 128>>>(N, cuda_p, cuda_Ap, cuda_out);
        hipMemcpy(&alpha, cuda_out, sizeof(double), hipMemcpyDeviceToHost);
        alpha = rr / alpha;

        xrIterate<<<128,128>>>(N, cuda_solution, cuda_p, cuda_r, cuda_Ap, alpha);

        beta = rr;

        dot<<<128, 128>>>(N, cuda_r, cuda_r, cuda_out);
        hipMemcpy(&rr, cuda_out, sizeof(double), hipMemcpyDeviceToHost);

        if (std::sqrt(rr/rr0) < 1e-6) break;

        beta = rr / beta;

        vecIterate<<<128, 128>>>(N, cuda_p, cuda_r, cuda_p, beta);
        hipDeviceSynchronize();

        if (iters > 10000) break;  // solver didn't converge
        ++iters;
    }

    hipMemcpy(solution, cuda_solution, N * sizeof(double), hipMemcpyDeviceToHost);


    free(p);
    free(r);
    free(Ap);
    hipFree(cuda_csr_rowoffsets);
    hipFree(cuda_csr_colindices);
    hipFree(cuda_csr_values);
    hipFree(cuda_p);
    hipFree(cuda_r);
    hipFree(cuda_Ap);
    hipFree(cuda_solution);
    hipFree(cuda_out);

}



/** Solve a system with `points_per_direction * points_per_direction` unknowns */
void solve_system(size_t points_per_direction) {

    size_t N = points_per_direction * points_per_direction; // number of unknows to solve for

    //std::cout << "Solving Ax=b with " << N << " unknowns." << std::endl;

    //
    // Allocate CSR arrays.
    //
    // Note: Usually one does not know the number of nonzeros in the system matrix a-priori.
    //       For this exercise, however, we know that there are at most 5 nonzeros per row in the system matrix, so we can allocate accordingly.
    //
    int *csr_rowoffsets =    (int*)malloc(sizeof(double) * (N+1));
    int *csr_colindices =    (int*)malloc(sizeof(double) * 5 * N);
    double *csr_values  = (double*)malloc(sizeof(double) * 5 * N);

    //
    // fill CSR matrix with values
    //
    generate_fdm_laplace(points_per_direction, csr_rowoffsets, csr_colindices, csr_values);

    //
    // Allocate solution vector and right hand side:
    //
    double *solution = (double*)malloc(sizeof(double) * N);
    double *rhs      = (double*)malloc(sizeof(double) * N);
    std::fill(rhs, rhs + N, 1);

    //
    // Call Conjugate Gradient implementation (CPU arrays passed here; modify to use GPU arrays)
    //
    Timer timer;
    double time;
    timer.reset();
    conjugate_gradient(N, csr_rowoffsets, csr_colindices, csr_values, rhs, solution);
    time = timer.get();
    std::cout << time;

    //
    // Check for convergence:
    //
    double residual_norm = relative_residual(N, csr_rowoffsets, csr_colindices, csr_values, rhs, solution);
    //std::cout << "Relative residual norm: " << residual_norm << " (should be smaller than 1e-6)" << std::endl;

    free(solution);
    free(rhs);
    free(csr_rowoffsets);
    free(csr_colindices);
    free(csr_values);

}


int main(int argc, char *argv[]) {

    solve_system(std::atoi(argv[1])); // solves a system with 100*100 unknowns
    return EXIT_SUCCESS;
}
