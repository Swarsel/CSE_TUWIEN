#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "timer.hpp"
#include "cuda_errchk.hpp"   // for error checking of CUDA calls

__global__
void transpose(double *A, int N)
{
  int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int total_threads = blockDim.x * gridDim.x;

  for (unsigned int i = thread_id; i < N*N; i += total_threads) {
      int r = i / N;
      int c = i % N;

      if (r < c) {
         double temp = A[N * r + c];
         A[N * r + c] = A[N * c + r];
         A[N * c + r] = temp;
      }
   }
}


void print_A(double *A, int N)
{
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; ++j) {
      std::cout << A[i * N + j] << ", ";
    }
    std::cout << std::endl;
  }
}

int main(void)
{
  double bw;
  std::cout << "[";
  for (int n=6; n<13;n++) {
  int N = pow(2,n);

  double *A, *cuda_A;
  Timer timer;

  // Allocate host memory and initialize
  A = (double*)malloc(N*N*sizeof(double));

  for (int i = 0; i < N*N; i++) {
    A[i] = i;
  }

  //print_A(A, N);


  // Allocate device memory and copy host data over
  CUDA_ERRCHK(hipMalloc(&cuda_A, N*N*sizeof(double)));

  // copy data over
  CUDA_ERRCHK(hipMemcpy(cuda_A, A, N*N*sizeof(double), hipMemcpyHostToDevice));

  // wait for previous operations to finish, then start timings
  CUDA_ERRCHK(hipDeviceSynchronize());
  timer.reset();

  // Perform the transpose operation
  transpose<<<(N+255)/256, 256>>>(cuda_A, N);

  // wait for kernel to finish, then print elapsed time
  CUDA_ERRCHK(hipDeviceSynchronize());
  double elapsed = timer.get();
  //std::cout << std::endl << "Time for transpose: " << elapsed << std::endl;
  //std::cout << "Effective bandwidth: " << (2*N*N*sizeof(double)) / elapsed * 1e-9 << " GB/sec" << std::endl;
  //std::cout << std::endl;
   bw = (2*N*N*sizeof(double)) / elapsed * 1e-9;
  if (N!=4096) std::cout << bw << ", ";

  // copy data back (implicit synchronization point)
  CUDA_ERRCHK(hipMemcpy(A, cuda_A, N*N*sizeof(double), hipMemcpyDeviceToHost));

  //print_A(A, N);

  CUDA_ERRCHK(hipFree(cuda_A));
  free(A);

  CUDA_ERRCHK(hipDeviceReset());  // for CUDA leak checker to work
}
  std::cout << bw << "]";

  return EXIT_SUCCESS;
}
