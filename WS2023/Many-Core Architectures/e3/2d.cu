#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <iostream>
#include "timer.hpp"
#include "cuda_errchk.hpp"   // for error checking of CUDA calls

__global__
void transpose(double *A)
{
  __shared__ float tile[16][16];

  int x = blockIdx.x * 16 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  int width = gridDim.x * 16;

  for (int j = 0; j < 16; j += 8)
     tile[threadIdx.y+j][threadIdx.x] = A[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * 16 + threadIdx.x;
  y = blockIdx.x * 16 + threadIdx.y;

  for (int j = 0; j < 16; j += 8)
     A[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}


void print_A(double *A, int N)
{
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; ++j) {
      std::cout << A[i * N + j] << ", ";
    }
    std::cout << std::endl;
  }
}

int main(void)
{
  double bw;
  std::cout << "[";
  for (int n=6; n<13;n++) {
  int N = pow(2,n);

  dim3 dimGrid(N/16, N/16, 1);
  dim3 dimBlock(16, 8, 1);

  double *A, *cuda_A;

  Timer timer;

  // Allocate host memory and initialize
  A = (double*)malloc(N*N*sizeof(double));

  for (int i = 0; i < N*N; i++) {
    A[i] = i;
  }

  // print_A(A, N);


  // Allocate device memory and copy host data over
  CUDA_ERRCHK(hipMalloc(&cuda_A, N*N*sizeof(double)));

  // copy data over
  CUDA_ERRCHK(hipMemcpy(cuda_A, A, N*N*sizeof(double), hipMemcpyHostToDevice));

  // wait for previous operations to finish, then start timings
  CUDA_ERRCHK(hipDeviceSynchronize());

  timer.reset();

  // Perform the transpose operation
  transpose<<<dimGrid, dimBlock>>>(cuda_A);

  // wait for kernel to finish, then print elapsed time
  CUDA_ERRCHK(hipDeviceSynchronize());
  double elapsed = timer.get();

  //std::cout << std::endl << "Time for transpose: " << elapsed << std::endl;
  //std::cout << "Effective bandwidth: " << (2*N*N*sizeof(double)) / elapsed * 1e-9 << " GB/sec" << std::endl;
  bw = (2*N*N*sizeof(double)) / elapsed * 1e-9;
  //std::cout << N << ", " << elapsed << ", " << bw << std::endl;

  if (N!=4096) std::cout << bw << ", ";

  // copy data back (implicit synchronization point)
  CUDA_ERRCHK(hipMemcpy(A, cuda_A, N*N*sizeof(double), hipMemcpyDeviceToHost));

  // print_A(A, N);

  CUDA_ERRCHK(hipFree(cuda_A));
  free(A);

  CUDA_ERRCHK(hipDeviceReset());  // for CUDA leak checker to work
  }
  std::cout << bw << "]";
  return EXIT_SUCCESS;
}
